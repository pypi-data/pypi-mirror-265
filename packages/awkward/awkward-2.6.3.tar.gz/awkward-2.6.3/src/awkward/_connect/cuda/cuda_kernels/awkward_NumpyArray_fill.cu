#include "hip/hip_runtime.h"
// BSD 3-Clause License; see https://github.com/scikit-hep/awkward-1.0/blob/main/LICENSE

template <typename T, typename C>
__global__ void
awkward_NumpyArray_fill(
    T* toptr,
    int64_t tooffset,
    const C* fromptr,
    int64_t length,
    uint64_t invocation_index,
    uint64_t* err_code) {
  if (err_code[0] == NO_ERROR) {
    int64_t thread_id = blockIdx.x * blockDim.x + threadIdx.x;
    if (thread_id < length) {
      toptr[tooffset + thread_id] = (T)fromptr[thread_id];
    }
  }
}
