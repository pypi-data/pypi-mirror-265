#include "hip/hip_runtime.h"
// BSD 3-Clause License; see https://github.com/scikit-hep/awkward-1.0/blob/main/LICENSE

// BEGIN PYTHON
// def f(grid, block, args):
//     (nextcarry, nextparents, parents, size, length, invocation_index, err_code) = args
//     scan_in_array = cupy.zeros(length * size, dtype=cupy.int64)
//     cuda_kernel_templates.get_function(fetch_specialization(['awkward_RegularArray_reduce_nonlocal_preparenext_a', nextcarry.dtype, nextparents.dtype, parents.dtype]))(grid, block, (nextcarry, nextparents, parents, size, length, scan_in_array, invocation_index, err_code))
//     scan_in_array = cupy.cumsum(scan_in_array)
//     cuda_kernel_templates.get_function(fetch_specialization(['awkward_RegularArray_reduce_nonlocal_preparenext_b', nextcarry.dtype, nextparents.dtype, parents.dtype]))(grid, block, (nextcarry, nextparents, parents, size, length, scan_in_array, invocation_index, err_code))
// out["awkward_RegularArray_reduce_nonlocal_preparenext_a", {dtype_specializations}] = None
// out["awkward_RegularArray_reduce_nonlocal_preparenext_b", {dtype_specializations}] = None
// END PYTHON

template <typename T, typename C, typename U>
__global__ void
awkward_RegularArray_reduce_nonlocal_preparenext_a(
    T* nextcarry,
    C* nextparents,
    const U* parents,
    int64_t size,
    int64_t length,
    int64_t* scan_in_array,
    uint64_t invocation_index,
    uint64_t* err_code) {
if (err_code[0] == NO_ERROR) {
    int64_t thready_id = blockIdx.x * blockDim.x + threadIdx.x;
    int64_t len = length * size;
    if (thready_id < len) {
      scan_in_array[thready_id] = 1;
    }
  }
}

template <typename T, typename C, typename U>
__global__ void
awkward_RegularArray_reduce_nonlocal_preparenext_b(
    T* nextcarry,
    C* nextparents,
    const U* parents,
    int64_t size,
    int64_t length,
    int64_t* scan_in_array,
    uint64_t invocation_index,
    uint64_t* err_code) {
  if (err_code[0] == NO_ERROR) {
    int64_t thready_id = (blockIdx.x * blockDim.x + threadIdx.x) / length;
    int64_t thread_id = (blockIdx.x * blockDim.x + threadIdx.x) % length;
    if (thread_id < length && thready_id < size) {
      nextcarry[scan_in_array[thready_id * length + thread_id] - 1] = thread_id * size + thready_id;
      nextparents[scan_in_array[thready_id * length + thread_id] - 1] = parents[thread_id] * size + thready_id;
    }
  }
}
